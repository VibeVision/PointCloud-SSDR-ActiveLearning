#include "hip/hip_runtime.h"

#include <stdio.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>



__global__ void NmDistanceKernel(int b,int n,const float * xyz,int m,const float * xyz2,float * result,int * result_i){
	const int batch=512;
	__shared__ float buf[batch*3];
	for (int i=blockIdx.x;i<b;i+=gridDim.x){
		for (int k2=0;k2<m;k2+=batch){
			int end_k=min(m,k2+batch)-k2;
			for (int j=threadIdx.x;j<end_k*3;j+=blockDim.x){
				buf[j]=xyz2[(i*m+k2)*3+j];
			}
			__syncthreads();
			for (int j=threadIdx.x+blockIdx.y*blockDim.x;j<n;j+=blockDim.x*gridDim.y){
				float x1=xyz[(i*n+j)*3+0];
				float y1=xyz[(i*n+j)*3+1];
				float z1=xyz[(i*n+j)*3+2];
				int best_i=0;
				float best=0;
				int end_ka=end_k-(end_k&3);
				if (end_ka==batch){
					for (int k=0;k<batch;k+=4){
						{
							float x2=buf[k*3+0]-x1;
							float y2=buf[k*3+1]-y1;
							float z2=buf[k*3+2]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (k==0 || d<best){
								best=d;
								best_i=k+k2;
							}
						}
						{
							float x2=buf[k*3+3]-x1;
							float y2=buf[k*3+4]-y1;
							float z2=buf[k*3+5]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+1;
							}
						}
						{
							float x2=buf[k*3+6]-x1;
							float y2=buf[k*3+7]-y1;
							float z2=buf[k*3+8]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+2;
							}
						}
						{
							float x2=buf[k*3+9]-x1;
							float y2=buf[k*3+10]-y1;
							float z2=buf[k*3+11]-z1;
							float d=x2*x2+y2*y2+z2*z2;
							if (d<best){
								best=d;
								best_i=k+k2+3;
							}
						}
					}
				}else{
					for (int k=0;k<end_ka;k